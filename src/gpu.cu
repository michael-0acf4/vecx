#include "hip/hip_runtime.h"
#include "gpu.cuh"
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void euclidean_norm_kernel(const float *data, uint64_t size, float *result)
{
    __shared__ float partial_sum[256];

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int local_tid = threadIdx.x;

    float sum = 0.0f;
    if (tid < size)
    {
        const float *fdata = (const float *)data;
        float val = fdata[tid];
        sum = val * val;
    }

    partial_sum[local_tid] = sum;
    __syncthreads();

    // fold (basic, within block)
    for (int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (local_tid < s)
            partial_sum[local_tid] += partial_sum[local_tid + s];
        __syncthreads();
    }

    // final result from thread 0 of block 0
    if (local_tid == 0 && blockIdx.x == 0)
    {
        atomicAdd(result, partial_sum[0]);
    }
}

void f32_norm_cuda(const float *data, uint64_t size, float *result)
{
    float *d_data = nullptr;
    hipMalloc(&d_data, size * sizeof(float));
    hipMemcpy((void *)d_data, data, size * sizeof(float), hipMemcpyHostToDevice);

    float *d_result = nullptr;
    hipMalloc(&d_result, sizeof(float));
    hipMemset(d_result, 0, sizeof(float));

    // Launch
    int threads = 256;
    int blocks = (size + threads - 1) / threads;
    euclidean_norm_kernel<<<blocks, threads>>>(d_data, size, d_result);

    float h_result = 0;
    hipMemcpy(&h_result, d_result, sizeof(float), hipMemcpyDeviceToHost);

    hipFree((void *)d_data);
    hipFree(d_result);

    *result = sqrtf(h_result);
}

extern "C" float f32_norm(const vecx *v)
{
    float r;
    f32_norm_cuda((const float *)v->data, v->size, &r);
    return r;
}
