#include "hip/hip_runtime.h"
#include "gpu.cuh"
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void euclidean_norm_kernel(const float *data, uint64_t size, float *result)
{
    extern __shared__ float partial_sum[];

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int local_tid = threadIdx.x;

    float sum = 0.0f;
    if (tid < size)
    {
        float val = data[tid];
        sum = val * val;
    }

    partial_sum[local_tid] = sum;
    __syncthreads();

    // fold (basic, within block)
    for (int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (local_tid < s)
            partial_sum[local_tid] += partial_sum[local_tid + s];
        __syncthreads();
    }

    // final result from thread 0
    if (local_tid == 0)
        atomicAdd(result, partial_sum[0]);
}

void f32_norm_cuda(const float *data, uint64_t size, float *result)
{
    float *d_data = nullptr;
    hipMalloc(&d_data, size * sizeof(float));
    hipMemcpy((void *)d_data, data, size * sizeof(float), hipMemcpyHostToDevice);

    float *d_result = nullptr;
    hipMalloc(&d_result, sizeof(float));
    hipMemset(d_result, 0, sizeof(float));

    // Launch
    int threads = 256;
    int blocks = (size + threads - 1) / threads;
    size_t shared_size = threads * sizeof(float);
    euclidean_norm_kernel<<<blocks, threads, shared_size>>>(d_data, size, d_result);
    hipDeviceSynchronize();

    float h_result = 0;
    hipMemcpy(&h_result, d_result, sizeof(float), hipMemcpyDeviceToHost);

    hipFree((void *)d_data);
    hipFree(d_result);

    *result = sqrtf(h_result);
}

extern "C" float f32_norm(const vecx *v)
{
    float r;
    f32_norm_cuda((const float *)v->data, v->size, &r);
    return r;
}
