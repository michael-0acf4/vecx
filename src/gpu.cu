#include "hip/hip_runtime.h"
#include "gpu.cuh"
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void euclidean_norm_kernel(const vecx *v, float *result)
{
    __shared__ float partial_sum[256];

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int local_tid = threadIdx.x;

    float sum = 0.0f;
    if (tid < v->size)
    {
        const float *fdata = (const float *)v->data;
        float val = fdata[tid];
        sum = val * val;
    }

    partial_sum[local_tid] = sum;
    __syncthreads();

    // fold (basic, within block)
    for (int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (local_tid < s)
            partial_sum[local_tid] += partial_sum[local_tid + s];
        __syncthreads();
    }

    // final result from thread 0 of block 0
    if (local_tid == 0 && blockIdx.x == 0)
    {
        atomicAdd(result, partial_sum[0]);
    }
}

void f32_norm_cuda(const vecx *v, float *result)
{
    vecx *d_vec = nullptr;
    float *d_result = nullptr;
    float h_result = 0;

    hipMalloc(&d_vec, sizeof(vecx));
    hipMemcpy(d_vec, v, sizeof(vecx), hipMemcpyHostToDevice);

    hipMalloc(&d_result, sizeof(float));
    hipMemset(d_result, 0, sizeof(float));

    // Launch
    int threads = 256;
    int blocks = (v->size + threads - 1) / threads;
    euclidean_norm_kernel<<<blocks, threads>>>(d_vec, d_result);
    hipMemcpy(&h_result, d_result, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_vec);
    hipFree(d_result);

    *result = sqrtf(h_result);
}

extern "C" float f32_norm(const vecx *v)
{
    float r;
    f32_norm_cuda(v, &r);
    return r;
}
