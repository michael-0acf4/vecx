#include "hip/hip_runtime.h"
#include "gpu.cuh"
#include <hip/hip_runtime.h>
#include <math.h>

template <typename T>
__device__ inline float maybe_dequantize(T value, const quant_params &)
{
    return static_cast<float>(value);
}
// Fallback
template <>
__device__ inline float maybe_dequantize<int8_t>(int8_t value, const quant_params &qparams)
{
    return qparams.scale *
           static_cast<float>(static_cast<int32_t>(value) - qparams.zero);
}

template <typename T>
__global__ void device_euclidean_norm_kernel(const T *data, uint64_t size, quant_params qparams, float *result)
{
    extern __shared__ float partial_sum[];

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int local_tid = threadIdx.x;

    float sum = 0.0f;
    if (tid < size)
    {
        float val = maybe_dequantize(data[tid], qparams);
        sum = val * val;
    }

    partial_sum[local_tid] = sum;
    __syncthreads();

    // fold (basic, within block)
    for (int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (local_tid < s)
            partial_sum[local_tid] += partial_sum[local_tid + s];
        __syncthreads();
    }

    // final result from thread 0
    if (local_tid == 0)
        atomicAdd(result, partial_sum[0]);
}

template <typename T>
float f32_norm_host(const vecx *v)
{
    size_t type_size = vecx_type_size(v->dtype);
    T *d_data = nullptr;
    hipMalloc(&d_data, v->size * type_size);
    hipMemcpy((void *)d_data, v->data, v->size * type_size, hipMemcpyHostToDevice);

    float *d_result = nullptr;
    hipMalloc(&d_result, sizeof(float));
    hipMemset(d_result, 0, sizeof(float));

    // Launch
    int threads = 256;
    int blocks = (v->size + threads - 1) / threads;
    size_t shared_size = threads * type_size;
    device_euclidean_norm_kernel<<<blocks, threads, shared_size>>>(d_data, v->size, v->qparams, d_result);
    hipDeviceSynchronize();

    float h_result = 0;
    hipMemcpy(&h_result, d_result, sizeof(h_result), hipMemcpyDeviceToHost);

    hipFree((void *)d_data);
    hipFree(d_result);

    return sqrtf(h_result);
}

float f32_norm(const vecx *v)
{
    return v->dtype == FLOAT_32 ? f32_norm_host<float>(v) : f32_norm_host<int8_t>(v);
}

// CUDA context init often skew test duration without this trick
__global__ void init_kernel() {}
void init_device()
{
    init_kernel<<<1, 1>>>();
    hipDeviceSynchronize();
}
